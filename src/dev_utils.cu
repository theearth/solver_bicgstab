#include "dev_utils.h"

void init()
{
    hipSetDevice(0);
}

void copymat2dev (csr_t* dev, csr_t* host)
{
    dev->n = host->n;
    dev->nnz = host->nnz;

    CUDA_SAFE_CALL (hipMemcpy (dev->ia, host->ia, (dev->n+1)* sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL (hipMemcpy (dev->ja, host->ja, dev->nnz * sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL (hipMemcpy (dev->val, host->val, dev->nnz * sizeof(double), hipMemcpyHostToDevice));
}

void freemat (csr_t* dev)
{
    CUDA_SAFE_CALL (hipFree (dev->ia));
    CUDA_SAFE_CALL (hipFree (dev->ja));
    CUDA_SAFE_CALL (hipFree (dev->val));
}

csr_t* allocmat2dev (int nnz, int n)
{
    csr_t* dev = (csr_t*) malloc (sizeof(csr_t));
    void *ia = NULL, *ja = NULL, *val = NULL;
    CUDA_SAFE_CALL (hipMalloc ((void**)&ia, (n+1)*sizeof(int)));
    CUDA_SAFE_CALL (hipMalloc ((void**)&ja, nnz * sizeof(int)));
    CUDA_SAFE_CALL (hipMalloc ((void**)&val,nnz * sizeof(double)));
    dev->ia = (int*)ia;
    dev->ja = (int*)ja;
    dev->val = (double*)val;
    return dev;
}